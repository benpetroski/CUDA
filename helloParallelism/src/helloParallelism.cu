
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myKernel()
{
  printf("Hello, world from the device!\n");
}

int main()
{
  myKernel<<<1,1>>>();
  hipDeviceSynchronize();
}
