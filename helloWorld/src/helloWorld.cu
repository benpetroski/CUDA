
#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello(){
	printf("Hello World! I'm a thread in block %d\n", blockIdx.x);
}

int main(int argc, char **argv){
	//launch kernel
	hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
	//force the prints to flush
	hipDeviceSynchronize();

	return 0;
}
