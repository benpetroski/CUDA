#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define N (3*10)

//void saxpy_cpu(int n, float a, float *x, float *y){
//	for(int i=0; i<n; ++i)
//		y[i]=a*x[i]+y[i];
//}

__global__ void saxpy_gpu(int n, float a, float *x, float *y){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n)
		y[i] = a*x[i] + y[i];
}

int main(void){
	float *x, *y; //host copies
	float *d_x, *d_y; //device copies
	int size = N*sizeof(float);

	//Allocate space for device copies
	hipMalloc((void **)&d_x, size);
	hipMalloc((void **)&d_y, size);

	//Allocate space for host copies of x and y and setup input values
	x = (float *)malloc(size);
	random_floats(x, N);
	y = (float *)malloc(size);
	random_floats(y, N);

	//Copy input to device
	hipMemcpy(d_x, &x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, &y, size, hipMemcpyHostToDevice);

	saxpy_gpu<<<3, 10>>>(N, 2.0f, d_x, d_y);
	hipDeviceSynchronize();

	//Copy result back to host
	hipMemcpy(d_y, &y, size, hipMemcpyDeviceToHost);

	//Cleanup
	hipFree(d_x); hipFree(d_y);
	free(x); free(y);
	return 0;
}
