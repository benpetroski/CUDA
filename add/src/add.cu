#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 512
int main(void) {
	int *a, *b, *c;			// Host copies
	int *d_a, *d_b, *d_c;	// Device copies
	int size = N*sizeof(int); // Need integer the size of space

	// Allocate device copies of a, b, c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	random_ints(a, N);
	random_ints(b, N);

	// Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on the GPU and pass the parameters
	add<<<N, 1>>>(d_a, d_b, d_c);

	// Copy device results back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	printf("Result: %d", c);

	// Free memory!
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
